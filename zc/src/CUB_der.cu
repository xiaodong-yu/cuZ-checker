#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "CUB_ssim.h"
#include "CUB_der.h"
#include "matrix.hpp"

__global__ void derivatives(float *data, float *der, int r3, int r2, int r1, size_t order) 
{
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int bid = blockIdx.x;

    float base;
    int i, j;
    int h=bid*(16-order*2);
    double dx, dy, dz;
    
    int wsize = (r2-order*2)/(16-order*2) + ((r2-order*2)%(16-order*2)?1:0);
    int lsize = (r1-order*2)/(16-order*2) + ((r1-order*2)%(16-order*2)?1:0);
    
    static __shared__ float shared[16*16*16];

    for (int w=0; w<wsize*(16-order*2); w+=(16-order*2)){
        for (int l=0; l<lsize*(16-order*2); l+=(16-order*2)){
            for (i=0; i<16; i++){
                if ((h+i)<r3 && (l+tidx)<r1 && (w+tidy)<r2){
                    shared[i*16*16+tidy*16+tidx] = data[(h+i)*r1*r2+(w+tidy)*r1+(l+tidx)];
                }
                    
            }
            __syncthreads();                  

            for (i=0; i<(16-order*2); i++){
                if (tidx<(16-order*2) && tidy<(16-order*2)){
                    if ((h+i)<(r3-order*2) && (l+tidx)<(r1-order*2) && (w+tidy)<(r2-order*2)){
                        base = shared[(i+order)*16*16+(tidy+order)*16+tidx];
                        dx = (shared[(i+order)*16*16+(tidy+order)*16+tidx+order*2] - base)/2;
                        base = shared[(i+order)*16*16+tidy*16+tidx+order];
                        dy = (shared[(i+order)*16*16+(tidy+order*2)*16+tidx+order] - base)/2;
                        base = shared[i*16*16+(tidy+order)*16+tidx+order];
                        dz = (shared[(i+order*2)*16*16+(tidy+order)*16+tidx+order] - base)/2;
                        //if (bid==0)printf("index=%i,%e,%i,%i\n",i+2,dz,h+tidy,l+tidx);
                        //if (Data!=0.0) printf("ddata%i,%i,%i,%i,%i,%i=%e\n",w,l,bid,i,tidx,tidy,sqrt(dx*dx+dy*dy+dz*dz));
                        der[(h+i)*(r1-order*2)*(r2-order*2)+(w+tidy)*(r1-order*2)+(l+tidx)] = sqrt(dx*dx+dy*dy+dz*dz);
                        //der[(h+i)*(r1-order*2)*(r2-order*2)+(w+tidy)*(r1-order*2)+(l+tidx)] = Data;
                        //if (der[(w+i)*(r1-order*2)*(r2-order*2)+(h+tidy)*(r1-order*2)+(l+tidx)]!=0.0) printf("ddata%i=%e\n",(w+i)*(r1-order*2)*(r2-order*2)+(h+tidy)*(r1-order*2)+(l+tidx),der[(w+i)*(r1-order*2)*(r2-order*2)+(h+tidy)*(r1-order*2)+(l+tidx)]);
                    
                    }
                }
            }
            __syncthreads();                  
        }
    }

}

__global__ void auto_corr(float *data, float *autocor, int r3, int r2, int r1, float avg, size_t autosize) 
{
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int bid = blockIdx.x;

    float base, sum;
    int i, j;
    int h=bid*(16-autosize);
    double dx, dy, dz;
    
    int wsize = (r2-autosize)/(16-autosize) + ((r2-autosize)%(16-autosize)?1:0);
    int lsize = (r1-autosize)/(16-autosize) + ((r1-autosize)%(16-autosize)?1:0);
    
    extern __shared__ float shared[];
    float *bdata = shared;
    float *cor = &shared[blockDim.x * blockDim.y * 16];
    unsigned mask;

    for (int w=0; w<wsize*(16-autosize); w+=(16-autosize)){
        for (int l=0; l<lsize*(16-autosize); l+=(16-autosize)){
            for (i=0; i<16; i++){
                if ((h+i)<r3 && (l+tidx)<r1 && (w+tidy)<r2){
                    bdata[i*16*16+tidy*16+tidx] = data[(h+i)*r1*r2+(w+tidy)*r1+(l+tidx)];
                }
                    
            }
            __syncthreads();                  

            for (i=0; i<(16-autosize); i++){
                if (tidx<(16-autosize) && tidy<(16-autosize)){
                    if ((h+i)<(r3-autosize) && (l+tidx)<(r1-autosize) && (w+tidy)<(r2-autosize)){
                        mask = __activemask();
                        base = bdata[i*16*16+tidy*16+tidx];
                    
                        for (j=1; j<=autosize; j++){
                            sum = (bdata[(i+j)*16*16+(tidy+j)*16+tidx+j]-avg) * (base-avg);

                            for (int offset = warpSize/2; offset > 0; offset /= 2) 
                            sum += __shfl_down_sync(mask, sum, offset);

                            if (tidx==0) cor[blockDim.y*(j-1)+tidy] = sum;
                        }
                    }
                }
                __syncthreads(); 

                if (tidy<autosize){
                    if (tidx < (16-autosize) && (w+tidx)<(r2-autosize))
                    {
                        sum = cor[blockDim.y*tidy+tidx];
                        mask = __activemask();
                    } else sum = 0;

                    for (int offset = warpSize/2; offset > 0; offset /= 2) 
                        sum += __shfl_down_sync(mask, sum, offset);

                    if (tidx==0) autocor[gridDim.x*tidy+blockIdx.x] += sum;
                }
                __syncthreads();                  
            }
        }
    }
}
